#include <iostream>
#include <chrono>

#include "../problem.hpp"
#include "../io.hpp"

int main(int argc, char const *argv[]) {
    if (argc != 3) {
        cout << "Usage: " << argv[0] << " CITIES PATH" << endl;
        return 1;
    }

    cout.precision(17);

    // NOTE: We use floats since single-precision arithmetic is
    // much faster than double precision on GPUs.

    cout << "Loading cities from " << argv[1] << "..." << endl;
    auto cities = read_cities<double>(argv[1]);
    cout << "Loaded " << cities.size() << " cities" << endl;

    cout << "Loading path from " << argv[2] << "..." << endl;
    auto path = read_path(cities, argv[2]);
    if (!is_valid(path.begin(), path.end())) {
        cout << "Input path is not valid !";
    }

    cout << "Input path score = " << score(path) << endl;

    // // Copy cities to unified memory
    // cout << "Copying to unified memory..." << endl;
    // City<float>* cuda_path;
    // hipMallocManaged(&cuda_path, path.size()*sizeof(City<float>));
    // for (size_t i = 0; i < path.size(); i++) {
    //     cuda_path[i] = path[i];
    // }

    // int N = path.size() - 1;
    // int blockSize = 512;
    // int numBlocks = (N + blockSize - 1) / blockSize;

    // // Compute distances on GPU
    // float* distances_out;
    // hipMallocManaged(&distances_out, (path.size()-1)*sizeof(float));
    // auto t1 = chrono::high_resolution_clock::now();
    // distances_l1<<<blockSize, numBlocks>>>(distances_out, cuda_path, path.size());


    // // Wait for GPU to finish before accessing on host
    // hipDeviceSynchronize();
    // auto t2 = chrono::high_resolution_clock::now();

    // std::cout << "Delta t2-t1: " 
	//                   << std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count()
	// 		                << " nanoseconds" << std::endl;

    // // Compute distances on GPU
    // t1 = chrono::high_resolution_clock::now();
    // distances_l2<<<blockSize, numBlocks>>>(distances_out, cuda_path, path.size());

    // // Wait for GPU to finish before accessing on host
    // hipDeviceSynchronize();
    // t2 = chrono::high_resolution_clock::now();

    // std::cout << "Delta t2-t1: " 
	//                   << std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count()
	// 		                << " nanoseconds" << std::endl;


    // for (size_t i = 0; i < path.size(); i++) {
       // cout << distances_out[i] << endl;
    // }

//   // build k-d tree
//   kdt::KDTree<Point> kdtree(coords_points);

//   // build query
//   const Point query(coords_points[0]);
	
//   // k-nearest neigbors search example
//   const int k = 25;
//   const std::vector<int> knnIndices = kdtree.knnSearch(query, k);

  
//   // Build NN table [cityId] -> [NN0, NN1, ......, NNK]
//   int** nearest;
    
//   hipMallocManaged(&nearest, coords_points.size()*sizeof(int*));
//   for(unsigned int i = 0; i < coords_points.size(); ++i) {
//     int* neigh;
//     hipMallocManaged(&neigh, k*sizeof(int));

//     // build query
//     Point query(coords_points[i]);
//     // k-nearest neigbors search
//     const std::vector<int> knnIndices = kdtree.knnSearch(query, k+1); // k+1 because the first one is the point itself
//     for (unsigned int j = 1; j < k+1; ++j) {
//       neigh[j-1] = knnIndices[j];
//     }
//     nearest[i] = neigh;
    
//   }



//   // Two-Opt
//   int blockSize = 64;
//   int numBlocks = (path_size + blockSize - 1) / blockSize;

  
//   bool improved = true;
//     delta_t *result; // will contain the best move as a delta_t struct obj. 
//     hipMallocManaged(&result, (path_size-3)*sizeof(delta_t));
//     bool *filled; // tells wether a move that improves the score has been found or not
//     hipMallocManaged(&filled, (path_size-3)*sizeof(bool));
    
//     while(improved) {
//         improved = false;
//         two_opt_step<<<numBlocks, blockSize>>>(coords, result, path_array, path_size, nearest, filled); // after this step, results contains all the pairs that improve path 
//         // choose a move in results
//         // if a move is chosen, update path, set improved to true, compute/print new total_distance for debugging if necessary
//         // else => return;
	
// 	 // Wait for GPU to finish before accessing on host
// 	hipDeviceSynchronize();
//         for (int i = 0; i < (path_size-3); ++i){
//             if (filled[i]) {
//                 std::cout << result[i].delta;
//                 break;
//             }
//         }
       
//     }

    return 0;
}
