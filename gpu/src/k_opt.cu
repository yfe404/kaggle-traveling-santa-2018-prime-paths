#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

#include "problem.hpp"
#include "k_opt.hpp"
#include "knn.hpp"
#include "io.hpp"

// NOTE: We use floats since single-precision arithmetic is
// much faster than double precision on GPUs.
#define PRECISION float

// CPU + GPU Functions
// -------------------

template <typename T>
__host__ __device__
T distance_l1(Coord<T> a, Coord<T> b);

template <typename T>
__host__ __device__
T distance_l2(Coord<T> a, Coord<T> b);

template <typename T>
__host__ __device__
T two_opt_score(City<T>* path, int k, int l);

template <typename T>
__host__ __device__
void two_opt_results(City<T>* path, int path_size, int** neighbors_idxs, int n_neighbors, delta_t<T>* results, int index, int stride);

// CUDA Kernels
// ------------

// /!\ Make sure results is initialized to an array of {0, 0, 0}
template <typename T>
__global__
void two_opt_pass_gpu_kernel(City<T>* path, int path_size, int** neighbors_idxs, int n_neighbors, delta_t<T>* results) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    two_opt_results(path, path_size, neighbors_idxs, n_neighbors, results, index, stride);
}

// Host Code
// ---------

// CPU single-threaded 2-opt
template <typename T>
vector<City<T>> two_opt_pass_cpu(vector<City<T>> path, int k) {
     // Build NN table [cityId] -> [NN0, NN1, ......, NNK]
     auto neighbors_idxs = get_knn(path, k);
     int** gpu_neighbors_idxs;
     hipMallocManaged(&gpu_neighbors_idxs, path.size()*sizeof(int*));
 
     for (size_t i = 0; i < path.size(); ++i) {
         gpu_neighbors_idxs[i] = &neighbors_idxs[i][0];
     }

    //  auto neigh


    // Will contain the best move as a delta_t struct obj
    delta_t<T>* results = (delta_t<T>*)malloc((path.size()-3)*sizeof(delta_t<T>));
    for (size_t i = 0; i < path.size(); ++i) {
        results[i] = {0, 0, 0};
    }

    // vector<delta_t<T>> results(path.size(), {0, 0, 0});

    two_opt_results(&path[0], path.size(), neighbors_idxs, k, results, 0, 1);

    // TODO: Maximize profit and apply to new_path
    auto new_path(path);

    return new_path;
}

// GPU multi-threaded 2-opt
template <typename T>
vector<City<T>> two_opt_pass_gpu(vector<City<T>> path, int k) {
    // Build NN table [cityId] -> [NN0, NN1, ......, NNK]
    auto neighbors_idxs = get_knn(path, k);
    int** gpu_neighbors_idxs;
    hipMallocManaged(&gpu_neighbors_idxs, path.size()*sizeof(int*));

    for (size_t i = 0; i < path.size(); ++i) {
        gpu_neighbors_idxs[i] = &neighbors_idxs[i][0];
    }

    // Two-Opt
    int blockSize = 64;
    int numBlocks = (path.size() + blockSize - 1) / blockSize;

    // Will contain the best move as a delta_t struct obj
    delta_t<T>* results;
    hipMallocManaged(&results, (path.size()-3)*sizeof(delta_t<T>));
    for (size_t i = 0; i < path.size(); ++i) {
        results[i] = {0, 0, 0};
    }

    // Copy path to GPU
    // TODO: Use memcpy instead ?
    City<T>* gpu_path;
    hipMallocManaged(&gpu_path, path.size()*sizeof(City<T>));
    for (size_t i = 0; i < path.size(); ++i) {
        gpu_path[i] = path[i];
    }

    // Call GPU kernel
    auto start = chrono::steady_clock::now();
    two_opt_pass_gpu_kernel<<<numBlocks, blockSize>>>(
        gpu_path, path.size(), gpu_neighbors_idxs, k, results
    );
    hipDeviceSynchronize();

    auto finish = chrono::steady_clock::now();
    cout << "Time: " << chrono::duration_cast<chrono::duration<double> >(finish - start).count() << " seconds" << endl;

    
    // TODO: Maximize profit and apply to new_path
    auto new_path(path);

    return new_path;
}

int main(int argc, char const *argv[]) {
    if (argc != 3) {
        cout << "Usage: " << argv[0] << " CITIES PATH" << endl;
        return 1;
    }

    chrono::time_point<chrono::steady_clock> start;
    chrono::time_point<chrono::steady_clock> finish;
    cout.precision(17);

    cout << "Loading cities from " << argv[1] << "..." << endl;
    auto cities = read_cities<PRECISION>(argv[1]);
    cout << "Loaded " << cities.size() << " cities" << endl;

    cout << "Loading path from " << argv[2] << "..." << endl;
    auto path = read_path(cities, argv[2]);
    if (!is_valid(path.begin(), path.end())) {
        cout << "Input path is not valid !";
    }

    cout << "Input path score = " << score(path) << endl;

    cout << "2-opt pass (CPU)" << endl;
    start = chrono::steady_clock::now();
    auto new_path = two_opt_pass_cpu(path, 15);
    finish = chrono::steady_clock::now();
    cout << "Time: " << chrono::duration_cast<chrono::duration<double> >(finish - start).count() << " seconds" << endl;
    cout << "New score = " << score(new_path) << endl;

    cout << "2-opt pass (GPU)" << endl;
    start = chrono::steady_clock::now();
    new_path = two_opt_pass_gpu(path, 15);
    finish = chrono::steady_clock::now();
    cout << "Time: " << chrono::duration_cast<chrono::duration<double> >(finish - start).count() << " seconds" << endl;
    cout << "New score = " << score(new_path) << endl;

    // NOTE: I removed the while(improved) loop from the two_opt_pass function,
    // I think it's better to handle this outside (like here).

    write_path(new_path.begin(), new_path.end(), "k_opt_" + to_string(score(new_path)) + ".csv");
    return 0;
}
